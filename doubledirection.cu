#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <omp.h>

// Max registered calls
#define DDIO_MAX_FC 0xFF
// Max registered kernel calls
#define DDIO_MAX_KC 0xFF
// Max function unit stack size
#define MAX_ddio_fu 	512

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

typedef struct {

} ddio_event;

typedef enum {
	DDIO_PIPELINE_CALL,
	DDIO_PRE_SYNC_CALL,
	DDIO_POST_SYNC_CALL,
	DDIO_ASYNC_CALL,
	DDIO_PRE_SYNC_KERNEL_CALL,
	DDIO_POST_SYNC_KERNEL_CALL,
	DDIO_ASYNC_KERNEL_CALL
} DDIO_CALL_SYNC;

typedef struct {
	union {
	} registers;
} ddio_variable;

typedef enum {
	NATIVE_HOST_CALL,
	NATIVE_DEVICE_CALL,
	DDIO_supported_call_types
} DDIO_call_type;

typedef unsigned short int ddio_function;
typedef void **ddio_function_params;

typedef struct {
	dim3 grid;
	dim3 block;
} t_useThreads;

typedef struct {
	DDIO_call_type callType;
	DDIO_CALL_SYNC sync;
	t_useThreads useThreads;
	ddio_function ddio_call;
	void **params;
} ddio_call;

typedef void *(*DDIO_function_call)(ddio_function_params params);

typedef enum {
	DDIO_SUCCESS,
	DDIO_UNKNOWN_ERROR
} DDIO_ERROR_CODES;

typedef union {
	bool ready;
	DDIO_call_type type;
} ThreadUnit;

// Function unit
typedef struct {
	union {
		bool ready;
		unsigned short int fp;
		ThreadUnit *threadUnit;
		unsigned int numThreads;
		unsigned int availableThreads;
		bool updated;
	} registers;
	ddio_call stack[MAX_ddio_fu];
} ddio_FU;

ddio_FU ddio_function_unit;

__shared__ DDIO_function_call ddio_device_calls[DDIO_MAX_FC];
DDIO_function_call ddio_host_calls[DDIO_MAX_FC];
DDIO_function_call ddio_host_kernels[DDIO_MAX_KC];
__shared__ DDIO_function_call ddio_device_kernels[DDIO_MAX_KC];

//#define DDIO_call(f,)

// NOP function for only host specific functions like I/O
extern __device__ void DDIO_NOP(ddio_function_params params);
extern __host__ void DDIO_KERNEL_CALL(ddio_function_params params);

extern __host__ __device__ void ddio_function_call(ddio_call call);
extern __host__ __device__ void ddio_kernel_call(ddio_call call);
extern __host__ __device__ void ddio_sync_fu();
extern __host__ __device__ void ddio_fu_process_call();

// Pass params macro
#define _VA(threadIdx,...) { \
		if(threadIdx !== NULL) { \
		   void *_args[] = { __VA_ARGS__ }; \
   		   return (void **)_args; \
		} else { \
		   void *_args[] = { __VA_ARGS__, &threadIdx }; \
		   return (void **)_args; \
		} \
}

__device__ void DDIO_NOP(void **params) {
}

__host__ void DDIO_DEVICE_KERNEL_CALL(ddio_function_params params) {
	ddio_call call = (ddio_call) *params[0];
	(ddio_device_kernels[call.ddio_call]) <<<call.useThreads.grid,call.useThreads.block>>> ((void **)call.params);
}

__host__ void DDIO_push_tohost_call(ddio_function_params parmas) {
	ddio_call call = (ddio_call) *params[0];
	call.callType = DDIO_HOST_CALL;

}


__host__ __device__ void ddio_kernel_call(ddio_call call) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 350
	ddio_device_kernels<<<call.useThreads.grid,call.useThreads.block>>>[call.ddio_call](call.params);
#else
	DDIO_push_tohost_call();
#endif
#else
	dim3 threadIdx;

if(call.useThreads.block.z == 1 && call.useThreads.block.y == 1 && call.useThreads.block.x > 1) {
#pragma omp parallel for
	for (threadIdx.x = 0;threadIdx.x < call.useThreads.block.x;threadIdx.x++) {
		// TODO: pass throught threadIdx...
		(ddio_host_kernels[call.ddio_call])(call.params);
	}
} else if(call.useThreads.block.z == 1 && call.useThreads.block.y > 1 && call.useThreads.block.x > 1) {
#pragma omp parallel for
	for (threadIdx.y = 0;threadIdx.y < call.useThreads.block.y;threadIdx.y++) {
		#pragma omp parallel for
			for (threadIdx.x = 0;threadIdx.x < call.useThreads.block.x;threadIdx.x++) {
				// TODO: pass throught threadIdx...
				(ddio_host_kernels[call.ddio_call])(call.params);
		}
	}
} else if(call.useThreads.block.z > 1 && call.useThreads.block.y > 1 && call.useThreads.block.x > 1) {
#pragma omp parallel for
	for (threadIdx.z = 0;threadIdx.z < call.useThreads.block.z;threadIdx.z++) {
	#pragma omp parallel for
		for (threadIdx.y = 0;threadIdx.y < call.useThreads.block.y;threadIdx.y++) {
		#pragma omp parallel for
			for (threadIdx.x = 0;threadIdx.x < call.useThreads.block.x;threadIdx.x++) {
				// TODO: pass throught threadIdx...
				(ddio_host_kernels[call.ddio_call])(call.params);
			}
		}
	}
}
#endif
}
__host__ __device__ void ddio_function_call(ddio_call call) {
#ifdef __CUDA_ARCH__
	switch (call.sync) {
	case DDIO_PIPELINE_CALL:
		(ddio_device_calls[call.ddio_call])(call.params);
		break;
	case DDIO_PRE_SYNC_CALL:
		__syncthreads();
		(ddio_device_calls[call.ddio_call])(call.params);
		break;
	case DDIO_POST_SYNC_CALL:
		(ddio_device_calls[call.ddio_call])(call.params);
		__syncthreads();
		break;
	case DDIO_ASYNC_CALL:
		(ddio_device_calls[call.ddio_call])(call.params);
		break;
	}
#else
	switch (call.sync) {
		case DDIO_PIPELINE_CALL:
		(ddio_host_calls[call.ddio_call])(call.params);
		break;
		case DDIO_PRE_SYNC_CALL:
#pragma omp barier
#pragma omp single {
		(ddio_host_calls[call.ddio_call])(call.params);
}
		break;
		case DDIO_POST_SYNC_CALL:
#pragma omp single {
		(ddio_host_calls[call.ddio_call])(call.params);
}
#pragma omp barier
		break;
		case DDIO_ASYNC_CALL:
		(ddio_host_calls[call.ddio_call])(call.params);
		break;
	}
#endif
}

__host__ __device__ void ddio_sync_fu() {
#ifdef __CUDA_ARCH__
	// if(ddio_function_unit.registers.updated) {
	__syncthreads();
#pragma omp barrier
	// }
#else // HOST CALL FUNCTION
	//if (ddio_function_unit.registers.updated) {
	hipDeviceSynchronize();
#pragma omp barrier
	//}
#endif
}

__host__ __device__ void ddio_fu_process_call() {

}

__global__ void bitreverse(void *data) {
	__syncthreads();
}

int main(void) {
	return 0;
}
